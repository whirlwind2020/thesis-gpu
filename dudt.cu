#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hipfft/hipfft.h>

/* Calculate the time derivative of fluid with velocity
 * ux, uy, uz. The size of any dimension of any cube
 * is size.  */
public float* dudt(float* ux, float* uy,
                          float* uz, int size) {
  hipError_t lastError = hipSuccess;
  // move ux to GPU
  void* ux_gpu;
  lastError = hipMalloc(&ux_gpu, sizeof(ux));
  // gpuarrays++
  lastError = hipMemcpy(ux_gpu, ux, 
                size*size*size, hipMemcpyHostToDevice);

  // calculate fftderiv of ux
  
