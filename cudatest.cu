#include "hip/hip_runtime.h"
//nvcc -arch sm_21 -o test -run --ptxas-options="-v" -lcufft cudatest.cu
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include </usr/include/hipfft/hipfft.h>

__global__ void cuda_print() {
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int bz = blockIdx.z;

  int bdx = blockDim.x;
  int bdy = blockDim.y;
  int bdz = blockDim.z;
  int gdx = gridDim.x;
  int gdy = gridDim.y;
  int gdz = gridDim.z;

  printf("Thread id (%d, %d, %d) \n", tx, ty,tz);
  //printf("blockdim (%d, %d, %d) \n", bdx, bdy, bdz);
  printf("Block idx (%d, %d, %d) \n", bx, by, bz);
  //printf("griddim  (%d, %d, %d) \n", gdx, gdy, gdz);
  printf("=======================\n");
}

int main(int argc, char** argv) {
  // do some cuda testing 
  hipError_t res; 
  printf("entered \n");
  /*cuda_print<<<dim3(2,2,2), dim3(1,1,1)>>>();
  res = hipDeviceSynchronize();*/
  /*cuda_print<<<dim3(1,1,1), dim3(2,2,2)>>>();
  res = hipDeviceSynchronize();*/
  /*cuda_print<<<dim3(2,2), dim3(2,2)>>>();
  res = hipDeviceSynchronize();
  cuda_print<<<dim3(2,2,1), dim3(1,2,3)>>>();
  res = hipDeviceSynchronize();*/

  int n = 4;
  float* mat = (float*) malloc(n*n*n*sizeof(float));
  int i,j,k;
  for (i=0; i < n*n*n; i++) {
    *(mat+i) = i;//%n;
  }
  for (i=0; i<n; i++) {
    printf("======= x sheet %d =====\n", i);
    for (j=0; j<n; j++) {
      for (k=0; k<n; k++) {
        printf("%f ", *(mat+n*n*i+n*j+k));
      }
      printf("\n\n");
    }
  }

  
  float* gpu_in;
  hipfftComplex* gpu_out;
  hipMalloc(&gpu_in, n*n*n*sizeof(float));
  hipMalloc(&gpu_out, n*n*(n/2+1)*sizeof(hipfftComplex));
  hipfftComplex* fft_out = (hipfftComplex*) malloc(n*n*(n/2+1)*sizeof(hipfftComplex));

  hipMemcpy(gpu_in, mat, n*n*n*sizeof(float), hipMemcpyHostToDevice);
  hipfftHandle plan;
  hipfftPlan3d(&plan, n,n,n, HIPFFT_R2C);
  hipfftExecR2C(plan, gpu_in, gpu_out);
  //hipfftDestroy(plan);

  
  hipMemcpy(fft_out, gpu_out, n*n*(n/2+1)*sizeof(hipfftComplex), hipMemcpyDeviceToHost);
  for (i=0; i<n; i++) {
    printf("======= x sheet %d =====\n", i);
    for (j=0; j<n; j++) {
      for (k=0; k<n/2+1; k++) {
        printf("%f ", ((hipfftComplex)*(fft_out+n*n*i+n*j+k)).x);
        printf("+%fi ", ((hipfftComplex)*(fft_out+n*n*i+n*j+k)).y);
      }
      printf("\n\n");
    }
  }

  hipfftPlan3d(&plan, n,n,n, HIPFFT_C2R);
  hipfftExecC2R(plan, gpu_out, gpu_in);
  
  hipMemcpy(mat, gpu_in, n*n*n*sizeof(float), hipMemcpyDeviceToHost);
  for (i=0; i<n; i++) {
    printf("======= x sheet %d =====\n", i);
    for (j=0; j<n; j++) {
      for (k=0; k<n; k++) {
        printf("%f ", (*(mat+i*n*n+j*n+k))/(n*n*n));
      }
      printf("\n\n");
    }
  }

}
 
